
#include <hip/hip_runtime.h>
#include <stdio.h>

// TMC Faster
__constant__ int mapping[20] = {0, -1, 3, -1, -1, -1, 2, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, 1};

// Thread i will score genome[i*seqlength] to genome[i*seqlength+(seqlength-1)]
__global__ void scoreReads(char* genome, int seqLength, int order, float* model, float* scores) {
   int i = blockIdx.x;  // Thread identifier, assign to i
   int j = threadIdx.x;
   // Keep scores in shared memory
   extern __shared__ float kmer_scores[];  // Call this with [lengths[i] / order + 1];

   //if (i ==0) printf("%s\n", genome);
   // Start spot
   int seqspot = i*seqLength;
   int startspot, stopspot;
   if (j == 0) {
      startspot = seqspot;
      stopspot = startspot+(order-1);
   }
   else{
      startspot = seqspot+(j-1);
      stopspot = startspot+order;
   }
   //printf("Block %d Thread %d Startspot %d Stopspot %d\n", i, j, startspot, stopspot);
   // Quick loop, check for n's
   // Actually, decided to inline it rather than loop twice.
   int a;
   bool nFlag = false;
   int mapVal = 0;
   for (a = startspot; a < stopspot; a++) {
      //if (j == 0 && i == 0) printf("%d %d\n", startspot, a);
      //if (j == 0 && i == 0) printf("%c\n", genome[a]);
      if (genome[a] == 'N') { 
         //if (i == 0) printf("FOUND N, BREAKING.\n");
         kmer_scores[j] = 0;
         nFlag = true;
         break;
      }
      else 
         mapVal = 4*mapVal + mapping[(int)genome[a]-65];
   }
   if (j == 1 && i == 0) printf("\n");
   if (!nFlag) {
      if (j == 0) {mapVal += pow(4.0, 9.0);}
      kmer_scores[j] = model[mapVal]; // Illegal here
      //if (i == 0) printf("Thread: %d  Mapval: %d  Score: %f\n", j, mapVal, kmer_scores[j]);
    }
   __syncthreads();

    /////// TMC TAKE OUT LATER
   /*if (j == 0) {
      int m;
      float tmpscore=0;
      for (m = seqspot; m < seqspot+seqLength-order+1+1; m++) {
         if (i == 0) printf("%d: Score: %f  New Partial Score: %f\n", m+8, kmer_scores[m-seqspot], tmpscore); 
         tmpscore += kmer_scores[m-seqspot];
      }
      if (i == 0) printf("The score for sequence %d should be: %f\n", i, tmpscore);
     
   }
   __syncthreads();*/
   //////////////////////////


   // Do the addition in parallel as well.
   //if (j == 0 && i == 0) printf("Number: %d", seqLength-order+1);
   if ((j == 0) && (((seqLength-order+1+1) % 2 == 1))) {/*if (i == 0) printf("Thread %d adding %d (%f) and %d (%f)", j, j, kmer_scores[j], seqLength-order+1, kmer_scores[seqLength-order+1]);*/ kmer_scores[j] = kmer_scores[j] + kmer_scores[seqLength-order+1]; /*if (i == 0) printf(" to get: %f\n", kmer_scores[j]);*/ }
   int k = (seqLength-order+1+1)/2;
   while (k >= 1) {
      //if (i == 0 && j == 0) printf("k is %d\n", k);
      if (j < k) {
	 //if (i == 0) printf("k: %d Thread %d adding %d (%f) and %d (%f) to get: %f\n", k, j, j, kmer_scores[j], j+k, kmer_scores[j+k], kmer_scores[j]+kmer_scores[j+k]);
         kmer_scores[j] = kmer_scores[j] + kmer_scores[j+k]; // Illegal here
      }
      __syncthreads();
      if (k != 1 && k % 2 != 0 && j == 0) {/*printf("k: %d Thread %d adding %d (%f) and %d (%f) to get: %f\n", k, j, j, kmer_scores[j], k-1, kmer_scores[k-1], kmer_scores[j]+kmer_scores[k-1]);*/              kmer_scores[j] = kmer_scores[j] + kmer_scores[k-1]; }// Uneven split, would be left out otherwise.
      k /= 2;
   }

   // The first kmer_score is now the final.
   if (j == 0) {
      scores[i] = kmer_scores[0];
      //printf("Kernel score for sequence %d: %f\n", i, scores[i]);
   }
}


