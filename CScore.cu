#include "hip/hip_runtime.h"
/*
 *      Author: 	Vanessa Aguiar-Pulido
 *      			Postdoctoral Research Associate
 *      			Bioinformatics Research Group (BioRG)
 *      			Florida International University (FIU)
 *      			Miami, FL, USA
 *
 *     	Contact: 	vaguiarp@fiu.edu or vaguiarpulido@gmail.com
 */

#include "CScore.h"
#include <sys/time.h>

#include "scoreReads.h"
#include <math.h>

CScore::CScore() {


}

CScore::~CScore() {

}

float CScore::scoreRead(string read, int order, map<string, int> kmerMap, vector<float> model) {
	float score = 0.0;
	bool first = true;
	int mappedIndex = -1;
	string tmpKmer = read.substr(0, order); //From the beginning, take k characters;

	//We need to calculate the initial probabilities
	while(first) {
		try {
			mappedIndex = kmerMap.at(tmpKmer);
			score += model.at(mappedIndex);
			first = false;
		} catch (...) { //If there is an N in the initial k-mer, iterate past it
			tmpKmer.erase(tmpKmer.begin());
			tmpKmer.push_back((char)read.at(order));
			order++;
		}
	}

	//cout << "tmpKmer " << order << ": " << tmpKmer << "\n";

	tmpKmer.push_back((char)read.at(order)); //First (k+1)-mer

	for(int j=order+1; j<read.length(); j++) { //Calculate the score of a read
		//cout << "tmpKmer " << j << ": " << tmpKmer << "\n";
		try {
			mappedIndex = kmerMap.at(tmpKmer); //This will return the position of the kmer
			score += model.at(mappedIndex);
			//cout << "Partial score "<< j << ": " << score << "\n";
		} catch (...) {} //If there's an N, just skip it
		tmpKmer.erase(tmpKmer.begin());
		tmpKmer.push_back((char)read.at(j));
	}

	//cout << "tmpKmer " << read.length() << ": " << tmpKmer << "\n";

	//We need to add the last kmer
	try {
		mappedIndex = kmerMap.at(tmpKmer);
		score += model.at(mappedIndex);
	} catch (...) {} //If there's an N, just skip it

	//cout << "Score for read "<< read << ": " << score << "\n";

	return score;

}

void CScore::scoreModels(string modelsPath, string readsFileName, string outputFile, int order) {
	ifstream listFile, modelFile;
	ofstream scoreResults;
	string modelName="", modelFull="";
	vector<float> model;
	float value = 0.0;
	int index=-1;//, mappedIndex=-1;
	string tmpKmer="", tmpRead="";
	//float tmpScore=0.0;
	//Prepare to load the reads
	CSequences* reads = new CSequences(readsFileName);


        // GPU arrays
        int num_seq = reads->getSequences().size();
        int read_length = reads->getSequences()[0].size(); // TMC for now assuming same length
        int nucleotides = num_seq*read_length;
	float* cpu_scores = (float*) malloc(num_seq*sizeof(float));
 
        char* cpu_genome = (char*) malloc(nucleotides*sizeof(char));
        // data() does not work, have to copy manually for now.
        for (int i = 0; i < num_seq; i++) {
	   cout << "Sequence " << i << ": " << reads->getSequences()[i] << endl;
           for (int j = 0; j < read_length; j++) {
              cpu_genome[i*read_length+j] = reads->getSequences()[i][j];
           }
        }
        cout << "CPU GENOME: " << string(cpu_genome) << endl;

        char* gpu_genome;
        hipMalloc((void**) &gpu_genome, nucleotides*sizeof(char));
        hipMemcpy(gpu_genome, cpu_genome, nucleotides*sizeof(char), hipMemcpyHostToDevice); // TMC I know this works for vectors of ints, need to check vectors of strings

	//Prepare to get the list of possible kmers for a model
	CKmers* kmers = new CKmers(order);

        order++;
	//Get the full list of models
	if(modelsPath.compare(modelsPath.length()-1,1,"/") !=0) {
		modelsPath += "/";
	}
	string command = "ls "+ modelsPath +" > models.txt";
	system(command.c_str());

	//Open the file containing the names of the models
	listFile.open("models.txt");
	//cout << "Let's open the models file\n";

	if (listFile.is_open()) {

		while(getline(listFile,modelName)) { //Retrieve the name of the model
			modelFull = modelsPath + modelName;
			//cout << "ModelFull: " << modelFull << "\n";
			modelFile.open(modelFull.c_str()); //Open the file that contains the probabilities

			if (modelFile.is_open()) {
				try { //In case there's something in the model's folder that shouldn't be there
					int num_models = pow(4,order) + pow(4, order-1);
                                        float* cpu_model = (float*) malloc(num_models * sizeof(float));
					cout << "Model: " << modelName << "\n";
                                        int i=0;
					while(i < num_models && modelFile >> index >> value) {
                                                cpu_model[i] = value;
						//model.push_back(value); //Store the model values
						//cout << "Model value: " << value << "\n";
                                                i++;
					}
					//cout << "Model size: " << model.size() << "\n";
					//cout << "First element: " << model.data()[0] << "\n";
					float* gpu_model;
					hipMalloc((void**) &gpu_model, num_models*sizeof(float));
					hipMemcpy(gpu_model, cpu_model, num_models*sizeof(float), hipMemcpyHostToDevice);


           				float* gpu_scores;
					hipMalloc((void**) &gpu_scores, num_seq*sizeof(float));

					//For each read calculate the score for the model
					// Call with num_seq blocks of order threads.
					int num_kmers = read_length - order + 1 + 1;
					//printf("Calling kernel.\n");
					//cout << "Using " << num_seq << " blocks of " << num_kmers << " threads.  Shared memory contains " << num_kmers << " floats." << endl;
					scoreReads<<<num_seq, num_kmers, num_kmers*sizeof(float)>>>(gpu_genome, read_length, order, gpu_model, gpu_scores);
					hipDeviceSynchronize();
					//printf("Called kernel.\n");
					hipError_t error = hipGetLastError();
  if(error != hipSuccess)
  {
    // print the CUDA error message and exit
    printf("CUDA error: %s\n", hipGetErrorString(error));
    exit(-1);
  }

					hipMemcpy(cpu_scores, gpu_scores, num_seq*sizeof(float), hipMemcpyDeviceToHost);
					//if (cpu_scores[0] != cpu_scores[1])
                                        //   cout << "Warning: Sequence 0 has " << cpu_scores[0] << " and Sequence 1 has " << cpu_scores[1] << endl;
					hipFree(gpu_model);
					hipFree(gpu_scores);
					free(cpu_model);
					for(int i=0; i<reads->getSequences().size(); i++) {
						//tmpScore = this->scoreRead((string)reads->getSequences().at(i), order, kmers->getKmerList(), model);
						//cout << "Score for read "<< i << ": " << cpu_scores[i] << "\n";

						//Replace the score stored if the new score is higher
						// TMC for now removing, since we are only doing one model
						//cout << "Score for sequence " << i << " (press return to continue): " << cpu_scores[i] << endl;
						//int x;cin >> x;
						if(this->scores.size() < reads->getSequences().size()) {
							this->scores.push_back(cpu_scores[i]);
							this->modelNames.push_back((string)modelName.substr(0,modelName.find(".")));
						}
						else {
							if (cpu_scores[i] > this->scores.at(i)) {
								this->scores.at(i) = cpu_scores[i];
								this->modelNames.at(i) = modelName.substr(0,modelName.find("."));
							}
						}
			//			exit(1);
					} //End while scoring reads
				} catch(...) {}

				modelFile.close();
				//cout << "Model cleared." << endl;
				model.clear();
			} //End if model was loaded
		} //End while reading models

		//Write the final scores to a file
		scoreResults.open(outputFile.c_str());
		if (scoreResults.is_open()) {
			scoreResults << "Best score\tBest model\n";
			for(int i=0; i < this->scores.size(); i++) {
				scoreResults << this->scores.at(i) << "\t" << this->modelNames.at(i) << "\n";
			}
			scoreResults.close();
		}

		listFile.close();
	} //End if list of models was read

	reads->~CSequences();
	kmers->~CKmers();
}

/*
int main(int argc, char* argv[]) {

	//string pathToModels = "/scratch/giri_projects/vanessa/Azad/scripts/model_database/";
	//string pathToModels = "/Users/vanessa/Documents/Work/ResearchInProgress/BioRG/Metagenomics/smallList/";
	string pathToModels = "/Users/vanessa/Documents/Work/ResearchInProgress/BioRG/Metagenomics/signature_6order/";
	CScore* s = new CScore();
	struct timeval tv1, tv2;
	gettimeofday(&tv1, NULL);

	//for(int i=0; i<1000; i++) {
		//s->scoreModels(pathToModels,"test.fa","scores.txt",6);
	//}

	s->scoreModels(pathToModels,"test.fa","scores.txt",6);
	//s->scoreModels(pathToModels,"test.fa","scores.txt",8);

	gettimeofday(&tv2, NULL);
	double tm = (double) (tv2.tv_usec - tv1.tv_usec)/1000000 + (double) (tv2.tv_sec - tv1.tv_sec);
	cout << "Time taken in execution = " << tm << " seconds\n";

	return 0;
}
*/
